#include <hip/hip_runtime.h>

__global__ 
void square(float* d_out, float* d_in)
{
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

void your_square(int array_size, float* d_out, float* d_in)
{
    square<<<1, array_size >>> (d_out, d_in);
}